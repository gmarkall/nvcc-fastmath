
#include <hip/hip_runtime.h>

__global__ void cuda_Euler(const float * __restrict__ real, float *imag, float *output, const float ANGLE, const int LENGTH)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x,
  offset  = gridDim.x * blockDim.x;

  while(tid < LENGTH) {
    output[tid] = real[tid] * cosf(ANGLE) + imag[tid] * sinf(ANGLE);    
    tid += offset;
  }
}
